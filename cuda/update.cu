#include "hip/hip_runtime.h"
#include <lcutil.h>

__global__ void updateGPU(float *u0, float *u1, int NXPROB, int NYPROB, int N){
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < N)
		if(u0[i] != 0.0)
			u1[i] = u0[i] + 0.1 * (u0[i+NYPROB]+u0[i-NYPROB]-2*u0[i]) + 0.1 * (u0[i+1]+u0[i-1]-2*u0[i]);
}

extern "C" void update(float *u0, float *u1, int NXPROB, int NYPROB){
	float* dev_u0, *dev_u1;

	/* malloc device memory */
	CUDA_SAFE_CALL(hipMalloc((void**) &dev_u0, NXPROB * NYPROB * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &dev_u1, NXPROB * NYPROB * sizeof(float)));

	/* Copy from host memory to device memory */
	CUDA_SAFE_CALL(hipMemcpy(dev_u0, u0, NXPROB * NYPROB * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_u1, u1, NXPROB * NYPROB * sizeof(float), hipMemcpyHostToDevice));
	
	const int BLOCK_SIZE = 1024;
	dim3 dimBl(BLOCK_SIZE);  
	dim3 dimGr(FRACTION_CEILING((NXPROB * NYPROB), BLOCK_SIZE)); 
	
	/* Update */
	updateGPU<<<dimGr, dimBl>>>(dev_u0, dev_u1, NXPROB, NYPROB, (NXPROB * NYPROB));
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	/* Copy from device memory to host memory */
	CUDA_SAFE_CALL(hipMemcpy(u0, dev_u0, NXPROB * NYPROB * sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL(hipMemcpy(u1, dev_u1, NXPROB * NYPROB * sizeof(float), hipMemcpyDeviceToHost) );

	/* Free device memory */
	CUDA_SAFE_CALL(hipFree(dev_u0));
	CUDA_SAFE_CALL(hipFree(dev_u1));
}

